#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

#include "caffe/layers/swish_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SwishForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = (1. / (1. + exp(-in[index])) ) * x;
  }
}

template <typename Dtype>
void SwishLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  SwishForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void SwishBackward(const int n, const Dtype* in_diff,
    const Ddype* bottom_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype sigmoid_x = (1. / (1. + exp(-bottom_data[index])) ) * bottom_data[index];
    out_diff[index] = in_diff[index] * (sigmoid_x * (1 - sigmoid_x) * bottom_data[index] + sigmoid_x);
  }
}

template <typename Dtype>
void SwishLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    SwishBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SwishLayer);


}  // namespace caffe
