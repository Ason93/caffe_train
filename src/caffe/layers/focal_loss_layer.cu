#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/focal_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void focalSoftmaxLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* label, Dtype* loss,
          const int num, const int dim, const int spatial_dim,
          const bool has_ignore_label_, const int ignore_label_,
          Dtype* counts, float gamma, float alpha) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);
    if (has_ignore_label_ && label_value == ignore_label_) {
      loss[index] = 0;
      counts[index] = 0;
    } else {
      Dtype prob_a = prob_data[n * dim + label_value * spatial_dim + s];
      Dtype b = powf(1- prob_a, gamma);
      loss[index] = -log(max(prob_a,
                      Dtype(FLT_MIN))) * b * alpha;
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void focalSoftmaxWithLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
    softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* label = bottom[1]->gpu_data();
    const int dim = prob_.count() / outer_num_;
    const int nthreads = outer_num_ * inner_num_;
    // Since this memory is not used for anything until it is overwritten
    // on the backward pass, we use it here to avoid having to allocate new GPU
    // memory to accumulate intermediate results in the kernel.
    Dtype* loss_data = bottom[0]->mutable_gpu_diff();
    // Similarly, this memory is never used elsewhere, and thus we can use it
    // to avoid having to allocate additional GPU memory.
    Dtype* counts = prob_.mutable_gpu_diff();
    // NOLINT_NEXT_LINE(whitespace/operators)
    focalSoftmaxLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, label, loss_data,
        outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts, gamma_, alpha_);
    Dtype loss;
    caffe_gpu_asum(nthreads, loss_data, &loss);
    Dtype valid_count = -1;
    // Only launch another CUDA kernel if we actually need the count of valid
    // outputs.
    if (normalization_ == LossParameter_NormalizationMode_VALID &&
        has_ignore_label_) {
      caffe_gpu_asum(nthreads, counts, &valid_count);
    }
    Dtype normalizer = LossLayer<Dtype>::GetNormalizer(
        normalization_, outer_num_, inner_num_, valid_count);
    top[0]->mutable_cpu_data()[0] = loss / normalizer;
    if (top.size() == 2) {
      top[1]->ShareData(prob_);
    }
}

template <typename Dtype>
__global__ void focalSoftmaxLossBackwardGPU(const int nthreads, const Dtype* top,
          const Dtype* label, const Dtype* prob_data, Dtype* bottom_diff, const int num, const int dim,
          const int spatial_dim, const bool has_ignore_label_,
          const int ignore_label_, Dtype* counts, float gamma, float alpha) {
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);
    Dtype focaldiff = 0;
    if (has_ignore_label_ && label_value == ignore_label_) {
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
      counts[index] = 0;
    } else {
      Dtype prob_a = prob_data[n * dim + label_value * spatial_dim + s];
      for(int c = 0; c < channels; ++c){
        if(c == label_value){
          Dtype diff_element = std::pow((1 - prob_a), gamma);
          Dtype diff_element_mutal = gamma *
                                    prob_a*log(max(prob_a,Dtype(FLT_MIN))) + prob_a -1;
          focaldiff = diff_element * diff_element_mutal * alpha;
        }else{
          Dtype pc = prob_data[n * dim + c * spatial_dim + s];
          Dtype diff_element = std::pow((1 - prob_a), gamma -1)*pc;
          Dtype diff_element_mutal =  1 - prob_a - gamma *
                                   prob_a*log(max(prob_a,Dtype(FLT_MIN)));
          focaldiff = diff_element * diff_element_mutal * alpha;
        }
        bottom_diff[n * dim + c * spatial_dim + s] = focaldiff;
      }
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void focalSoftmaxWithLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
                << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* label = bottom[1]->gpu_data();
    const int dim = prob_.count() / outer_num_;
    const int nthreads = outer_num_ * inner_num_;
    Dtype* counts = prob_.mutable_gpu_diff();
    focalSoftmaxLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, top_data, label, prob_data, bottom_diff,
        outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts, gamma_, alpha_);

    Dtype valid_count = -1;
    if (normalization_ == LossParameter_NormalizationMode_VALID &&
        has_ignore_label_) {
      caffe_gpu_asum(nthreads, counts, &valid_count);
    }
    Dtype normalizer = LossLayer<Dtype>::GetNormalizer(
        normalization_, outer_num_, inner_num_, valid_count);
    const Dtype loss_weight = top[0]->cpu_diff()[0] / normalizer;
    caffe_gpu_scal(prob_.count(), loss_weight , bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(focalSoftmaxWithLossLayer);

}  // namespace caffe
